#include "hip/hip_runtime.h"
#include <stdio.h>

#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

__global__
void squareKernel(int *data, int N)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i < N)
	{
		data[i] = i * i;
	}
}

int main(int argc, char **argv)
{
	int *h_data;
	int *d_data;
	int n = 1000;
	int sum = 0;

	hipHostAlloc(&h_data, n * sizeof(int), hipHostMallocPortable);
	hipMalloc(&d_data, n * sizeof(int));

	dim3 block(512);
	dim3 grid((n + block.x - 1) / block.x);

	//hipMemcpy(d_data, h_data, n * sizeof(int), hipMemcpyHostToDevice);
	squareKernel<<<grid, block>>>(d_data, n);
	hipMemcpy(h_data, d_data, n * sizeof(int), hipMemcpyDeviceToHost);

	for(int j=0;j<n;j++)
	{
		sum = sum + h_data[j];
	}
	
	printf("square of %d = %d\n", n-2, h_data[n-2]);
	printf("square of %d = %d\n", n-1, h_data[n-1]);
	printf("sum = %d\n", sum);
	return 0;
}